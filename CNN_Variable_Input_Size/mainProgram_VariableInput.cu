#include "hip/hip_runtime.h"


#include"Header.h"
#include"cudaKernels.cu"

//============================================================================================================================================================
// This program implement the CUDA code required to train a deep residual convolution network using a variable input size. The input here is the 3 RGB input
// channels of the training images. The network is trained using a set of predefined sizes, and in each iteration one of these sizes is randomly chosen and 
// and used to train the network. The sizes of the input and output channels for all conv layers, pooling layers, and batch normalization layers will also
// change according to the changes in the input size. To simplify the implementation, all images in a single batch use the same randomly chosen input size, 
// and therefore the GPU buffers required to hold the input and output channels of all layers will be allocated for the maximum input size. This restricts
// the maximum value of the input size based on the available GPU memory. On the other hand, the conv layer cudnn.lib descriptors and core cuda kernels will be
// executed with multiple settings corresponding to the multiple predefined input sizes. The conv layer is implemented using the cudnn.lib 
// library provided by NVIDIA which implements different optimized versions of the conv layer that runs on fast NVIDIA GPUs. This program is implemented using
// Microsoft Visual Studio 2013. 
//
//============================================================================================================================================================



//============================================================================================================================================================
// The Operation_Mode variable defines the modes of operation for the main program. There are five modes of operation that are defined as follows:
// 
// Operation_Mode = TRAIN :- The main program performs training without inference. Training starts from random weights initialized using Kaiming 
//                  initialization method, and at the end of the training phase the network weights are saved into a text file.
// Operation_Mode = TRAIN_PLUS_INFERENCE :- The main program performs training and inference. Training starts from random weights initialized using Kaiming 
//                  initialization method, and at the end of the training phase the network weights are saved into a text file. Then the program then carries
//                  out multi-crop (or single-crop) inference on the test set.
// Operation_Mode = INTERRUPTED_TRAIN :- The main program resumes training after it has been interrupted. The training process will be resumed from the last
//                  saved copy of the network parameters, which can save days or even weeks of training. At the end of the training phase the network weights 
//                  are saved into a text file, and no inference is carries out.
// Operation_Mode = INTERRUPTED_TRAIN_PLUS_INFERENCE :- The main program resumes training after it has been interrupted, and then it performs inference on the 
//                  test set. The training process will be resumed from the last saved copy of the network parameters. At the end of the training phase the 
//                  network weights are saved into a text file.
// Operation_Mode = INFERENCE :- The main program performs inference only using a previously trained network. The program loads the weights of a trained 
//                  network, and the fixed means-variances calculated using the same network, and carries out inference on the test set. 
//
//============================================================================================================================================================


int Operation_Mode = TRAIN_PLUS_INFERENCE;


//============================================================================================================================================================
// Most of the parameters that define the settings of the program and the network structure are constant in the sense that they do not change during training.
// All such parameters are defined in the ConstantSettings.h file. The settings of the program can easily be changed by changing those parameters defined as 
// constant integers. The few floating point parameters that are defined here are the learning rate lr, the L2 weight decay parameter lmda. The learning rate
// is divided over the batch size to remind us to increase the learning rate with the same amount we increase the batch size to maintain the same effective
// learning rate. By dividing the learning rate by batch here, the SGD update equation implicitly divides lmda by BatchSize also, and therefore the effective
// lmda is 0.0005 in this case when the batch size is 100.
//============================================================================================================================================================


float lr = 0.1f / float(BatchSize);
float lmda = 0.05f;


//============================================================================================================================================================
// The program assumes that the training data, validation data, and and test data files that contain the input images and image labels are stored in a folder 
// where the the full directory path (full folder name) is stored in the DataFloder[] char array. The program assumes that there are 5 files to store each dataset.
// 3 files to store the 3 input RGB channels, one file to store the image labels, and one file to store the image dimensions (height and width). In the file that 
// stores the image dimensions the heights for all images should be stored first and then the widths for all images should stored second. The image labels are stored
// as integer values that reflect the class number. The RGB channels are stored as unsigned char buffers where each pixel can take an integer value between 0 and 255.
//
// The 5 file names used for the training dataset are :-
//
// trainRed.txt :- to hold the Red input channels for the training images.
// trainGreen.txt :- to hold the Green input channels for the training images.
// trainBlue.txt :- to hold the Blue input channels for the training images.
// trainlabels.txt :- to hold the image labels for the training images.
// CoordinatesTr.txt :- to hold the image dimensions for the training images.
//
// The 5 file names used for the validation dataset are :-
//
// valRed.txt :- to hold the Red input channels for the validation images.
// valGreen.txt :- to hold the Green input channels for the validation images.
// valBlue.txt :- to hold the Blue input channels for the validation images.
// vallabels.txt :- to hold the image labels for the validation images.
// CoordinatesVal.txt :- to hold the image dimensions for the validation images.
//
// If the validation set is also used as a test set as is usually the case for datasets such as ImageNet, then here is no need to include a third dataset defined
// as the test set. If there is a separate test set then the 5 file names used for the test dataset are :-
//
// testRed.txt :- to hold the Red input channels for the test images.
// testGreen.txt :- to hold the Green input channels for the test images.
// testBlue.txt :- to hold the Blue input channels for the test images.
// testlabels.txt :- to hold the image labels for the test images.
// CoordinatesTs.txt :- to hold the image dimensions for the test images.
//
// These file names can be changed of course in the  ReadFile() CPU thread that reads training and validation images to the main memory, at the start of the 
// inference phase in the main program when the input test image are read, and in the InitializeTrainingData() function that reads and initializes the buffers
// that hold the images labels and dimensions. However the way the 5 file5 setup shouldn't be changed without changing the code to accommodate such changes.
// Translating the .JPEG images into 3 files to hold the 3 RGB channels was done using MATLAB code which makes easier it to inspect the generated files. 
// Changing the code to directly read .JPEG can be done by adding extra code or by changing the functions that read the input images form the disk drive.
////============================================================================================================================================================


char DataFloder[] = "C:/FolderPath/FolderName/";


//============================================================================================================================================================
// The main program runs two CPU threads, where one reads the data from disk to memory and the other consumes data from memory. The following few parameters 
// are used to synchronize between the 2 (producer-consumer) CPU threads.
//============================================================================================================================================================


int slot = 0;
mutex mu;
std::condition_variable not_empty, not_full;


//============================================================================================================================================================
// Few auxiliary variables plus the random generator used to generate random numbers from different distributions.
//============================================================================================================================================================

float ErrorT, ErrorV, MSE = 0.0f;
int CountT, CountV;

random_device rd;
mt19937 gen(rd());

// InitializeTrainingData() should be changed to InitializeTrainingInferenceData
// slot should be changed to NumSlots
// CoordinatesTs should be changed into trainDimensions


//============================================================================================================================================================
// The ReadFile() function implements the CPU thread that reads data from the SSD drive to the main memory. The function uses the mutex variable mu, the 
// condition variable not_empty and not_full, and the integer variable slot to synchronize with the main CPU thread that consumes the data from the main 
// memory and use it to train the network. The image dimensions and labels are much smaller in size in comparison to the size of images themselves (the
// size of RGB input channels), and therefore they are read before the start of the training phase. This function only reads the input RGB channels one 
// segment at a time and put it in one half of the RGB buffers while the main CPU thread consumes the data from the other half. This thread reads the input
// RGB channels for the training dataset and for the validation dataset. 
//============================================================================================================================================================

void ReadFile(unsigned char *Red, unsigned char *Green, unsigned char *Blue, size_t *PartSize, int NParts, int VParts, int i0)
{
	FILE *in1, *in2, *in3, *in4, *in5, *in6;

	char FileName[128];

	strcpy_s(FileName, 128, DataFloder); strcat_s(FileName, 128, "trainRed.txt");
	fopen_s(&in1, FileName, "rb");
	strcpy_s(FileName, 128, DataFloder); strcat_s(FileName, 128, "trainGreen.txt");
	fopen_s(&in2, FileName, "rb");
	strcpy_s(FileName, 128, DataFloder); strcat_s(FileName, 128, "trainBlue.txt");
	fopen_s(&in3, FileName, "rb");
	strcpy_s(FileName, 128, DataFloder); strcat_s(FileName, 128, "valRed.txt");
	fopen_s(&in4, FileName, "rb");
	strcpy_s(FileName, 128, DataFloder); strcat_s(FileName, 128, "valGreen.txt");
	fopen_s(&in5, FileName, "rb");
	strcpy_s(FileName, 128, DataFloder); strcat_s(FileName, 128, "valBlue.txt");
	fopen_s(&in6, FileName, "rb");



	for (int i = i0; i < NParts*NumEpoch&& MSE < 100; i++)
	{
		int p = i%NParts;
		size_t altr = (i % 2)*(RGB_GPU_SIZE / 2);


		if (p == 0)
		{
			fseek(in1, long(0), SEEK_SET);
			fseek(in2, long(0), SEEK_SET);
			fseek(in3, long(0), SEEK_SET);
			fseek(in4, long(0), SEEK_SET);
			fseek(in5, long(0), SEEK_SET);
			fseek(in6, long(0), SEEK_SET);
		}
		//================================================================================

		unique_lock<mutex> locker(mu);

		while (slot == 2 && MSE < 100)
			not_full.wait(locker);

		if (p < NParts - VParts)
		{
			size_t numread1 = fread(Red + altr, sizeof(unsigned char), PartSize[p], in1);
			size_t numread2 = fread(Green + altr, sizeof(unsigned char), PartSize[p], in2);
			size_t numread3 = fread(Blue + altr, sizeof(unsigned char), PartSize[p], in3);
		}
		else
		{
			size_t numread4 = fread(Red + altr, sizeof(unsigned char), PartSize[p], in4);
			size_t numread5 = fread(Green + altr, sizeof(unsigned char), PartSize[p], in5);
			size_t numread6 = fread(Blue + altr, sizeof(unsigned char), PartSize[p], in6);
		}

		slot++;

		not_empty.notify_one();
		locker.unlock();

		//================================================================================
	}

	fclose(in1);
	fclose(in2);
	fclose(in3);
	fclose(in4);
	fclose(in5);
	fclose(in6);

}

//============================================================================================================================================================

//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%

//            The  Main Propgram 

//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%


int main()
{

	//=======================================================================================================================================================
	//
	// First, different types of return values for different cuda libraries and functions are defined. We used these return values lightly on the fly for 
	// debugging but they can be used intensively to spot the cause for error, which can be done by adding an if statement to each return value. However, 
	// because the cuda code for implementing deep CNNs is complex, all cuda kernels were implemented and tested individually, and therefore the main program 
	// didn't require intensive code for exception handling and debugging. This also makes the code easier to read. 
	// 
	//=======================================================================================================================================================

	hipError_t cudaStatus;
	hipblasStatus_t blasstatus;
	hiprandStatus_t hiprandStatus;

	hipdnnStatus_t dnnStatus;
	hipdnnHandle_t dnnHandle;
	dnnStatus = hipdnnCreate(&dnnHandle);

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	hipEvent_t start, stop;
	cudaStatus = hipEventCreate(&start);
	cudaStatus = hipEventCreate(&stop);



	//=======================================================================================================================================================
	//
	//         Initializing a cuda random generator.
	// 
	//=======================================================================================================================================================

	hiprandGenerator_t cuda_gen;
	hiprandStatus = hiprandCreateGenerator(&cuda_gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandStatus = hiprandSetPseudoRandomGeneratorSeed(cuda_gen, rd());



	//=======================================================================================================================================================	
	//
	// Initializing output files to store the network parameters, the per epoch results, and the BN fixed means and variances that are used in the inference stage. 
	//
	//=======================================================================================================================================================

	FILE *outResults, *outParam, *outMeansVariances, *outParamCopy, *inParam, *inMeansVariances;

	outResults = fopen("Results.txt", "wb");

	//=======================================================================================================================================================
	// If the Operation_Mode involves training Then Three output file are opened to store data. The outParam file pointer is used to store the parameters
	// of the trained network. The outParamCopy file pointer is used to store a copy of the network parameters (including a running average of the derivative
	// per parameters used during training by RMSprop) multiple times during training in case training is interrupted, training can be resumed from the last
	// stored copy, which is very importatnt when training lasts for days and weeks. The outMeansVariances file pointer is used to store the fixed means 
	// and variances of BN that are used in standard inferenece.
	//=======================================================================================================================================================

	if (Operation_Mode == TRAIN || Operation_Mode == INTERRUPTED_TRAIN || Operation_Mode == TRAIN_PLUS_INFERENCE || Operation_Mode == INTERRUPTED_TRAIN_PLUS_INFERENCE)
	{
		outParam = fopen("Output_param_final.txt", "wb");
		outParamCopy = fopen("Output_param_copy.txt", "wb");
		outMeansVariances = fopen("Output_MeansVariances.txt", "wb");
	}

	//=======================================================================================================================================================
	// In case training was interrupted the inParam file pointer is used to read the last stored copy of the network paramters, and resume training from 
	// that point.
	//=======================================================================================================================================================

	if (Operation_Mode == INTERRUPTED_TRAIN || Operation_Mode == INTERRUPTED_TRAIN_PLUS_INFERENCE)
	{
		inParam = fopen("Input_param_copy.txt", "rb");
	}

	//=======================================================================================================================================================
	// If the Operation_Mode includes inference only, then the inParam file pointer is used to load the parameters of a previously trained network, and the 	
	// inMeansVariances file pointer is used to load the fixed BN means and variances produced by the same network previously.  
	//=======================================================================================================================================================

	if (Operation_Mode == INFERENCE)
	{
		inParam = fopen("Input_param_final.txt", "rb");
		inMeansVariances = fopen("Input_MeansVariances.txt", "rb");
	}




	//=======================================================================================================================================================
	//
	// The input images are stored in three buffers Red, Green, and Blue, each storing one of the 3 RGB input channels for all images. Each buffer has 
	// a predefined size of RGB_GPU_SIZE bytes. Each buffer is divided into two equally sized parts where one CPU thread fills data into one part while 
	// another CPU thread (that executes the main program) consumes the data from the other part. Therefore the training data is divided into multiple 
	// segment if the total size of each of the RGB channels exceeds RGB_GPU_SIZE/2. The function InitializeTrainingData() allocates nd initializes the 
	// following data buffers.
	//
	//
	// d_HeightTr and d_WidthTr:-  GPU buffers stores the image heights and widths for all training\validation images and later for test images. 
	// d_StartTr:-  GPU buffer stores the first memory/file byte location for each image.
	// d_T :-       GPU buffer stores the image labels. 
	// NParts :- number of data segments in case the RGB total channel sizes for training+validation or test images exceeds RGB_GPU_SIZE/2.
	// VParts :- number of data segments in case the RGB total channel sizes for the validation images exceeds RGB_GPU_SIZE / 2.
	// PStart :-    buffer stores the image index of the first image in each data segment.  
	// StartPart :- buffer stores the first byte location for each data segment.
	// PartSize :-  buffer stores the size (number of bytes) for each data segemnt. 
	// 
	//=======================================================================================================================================================

	unsigned char *Red, *Green, *Blue;

	cudaStatus = hipHostAlloc(&Red, sizeof(unsigned char)*RGB_GPU_SIZE, hipHostMallocDefault);
	cudaStatus = hipHostAlloc(&Green, sizeof(unsigned char)*RGB_GPU_SIZE, hipHostMallocDefault);
	cudaStatus = hipHostAlloc(&Blue, sizeof(unsigned char)*RGB_GPU_SIZE, hipHostMallocDefault);


	unsigned int *d_HeightTr, *d_WidthTr;
	int *d_T, *PStart, NParts, VParts;
	size_t *d_StartTr, *PartSize, *StartPart;


	InitializeTrainingData(&d_HeightTr, &d_WidthTr, &d_StartTr, &d_T, &PStart, &PartSize, &StartPart, &NParts, &VParts, TrainSizeM + ValSizeM);




	//=======================================================================================================================================================
	//
	// The InitializeConvLayerParam() initializes the following variables which define the structure of the convolutional layers. Because the network is 
	// trained using a variable input size, some of the parameters have multiple sizes that correspond to multiple network input sizes. Such parameters 
	// are defined in the Var_Param c struct.
	//
	// CL :-    The number of convolutional layers.
	// P :-     a Var_Param c struct that includes member variables that define conv layer parameters with multiple sizes that correspond to multiple network input sizes.
	// P[k].IR[i] :- the height / width of a single square input channel for convolutional layer i and for the network input size k.
	// P[k].CR[i] :- the height / width of a single square output channel before maxpooling for convolutional layer i and for the network input size k.
    // P[k].SR[i] :- the height / width of a single square output channel after maxpooling for convolutional layer i and for the network input size k.
	// P[k].Xr[i], Xc[i] :- where P[k].Xr[i]*Xc[i] defines the total size of the input channels for conv layer i and for the network input size k.
	// P[k].Yr[i], Yc[i] :- where P[k].Xr[i]*Xc[i] defines the total size of the output channels for conv layer i and for the network input size k.	
	// WSize[i] :-     Number of weights in convolutional layer i.
	// InCh[i] :-      number of input channels for convolutional layer i.
	//
	// TEMP:-          temporary main memory buffer used to swap data between main memory and GPU memory.
	//
	//=======================================================================================================================================================

	int  Xc[CL],  Yc[CL], WSize[CL], InCh[CL];
	Var_Param P[NumWin];

	float *TEMP;

	InitializeConvLayerParam_Var(P, InCh, Xc, Yc, WSize, &TEMP);


	//=======================================================================================================================================================
	//
	// The InitialzeCuDNN() initializes the following cudnn.lib tensor, filter, and convolution descriptors and convolution algorithms for the convolutional 
	// layers. The tensor descriptors at each conv layer have multiple sizes because the network is trained using a variable input size.
	//
	// Desc_X[k][i] :-    Tensor descriptor of the input channels for conv layer i and for the network input size k.
	// Desc_Y[k][i] :-    Tensor descriptor of the output channels of conv layer i and for the network input size k.
	// Desc_Xs[k][i] :-   Tensor descriptor of the input channels of the residual connection at layer i and for the network input size k.
	// Desc_Ys[k][i] :-   Tensor descriptor of the output channels of the residual connection at layer i and for the network input size k.
	// Desc_W[i] :-       Filter descriptor of convolutional layer i.
	// Desc_Ws[i] :-      Filter descriptor of the residual connection at layer i.
	// Conv_Desc[i] :-    Convolution descriptor for convolutional layer i.
	// Conv_s_Desc[i] :-  Convolution descriptor for the residual connection at layer i.
	// FwdAlg[i] :-    Convolution algorithm to propagate the signal forward through convolutional layer i.
	// BwdDataAlg :-   Convolution algorithm to propagate the error signal backward through convolutional layer i.
	// BwdFilterAlg :- Convolution algorithm to propagate the error signal for the purpose of updating the parameters of convolutional layer i
	//
	//=======================================================================================================================================================

	hipdnnTensorDescriptor_t Desc_X[NumWin][CL], Desc_Y[NumWin][CL], Desc_Xs[NumWin][CL], Desc_Ys[NumWin][CL];
	hipdnnFilterDescriptor_t Desc_W[CL], Desc_Ws[CL];
	hipdnnConvolutionDescriptor_t Conv_Desc[CL], Conv_s_Desc;

	hipdnnConvolutionFwdAlgo_t FwdAlg[CL];
	hipdnnConvolutionBwdDataAlgo_t BwdDataAlg[CL];
	hipdnnConvolutionBwdFilterAlgo_t BwdFilterAlg[CL];

	InitialzeCuDNN_Var(Desc_X, Desc_Y, Desc_W, Conv_Desc, Desc_Xs, Desc_Ys, Desc_Ws, &Conv_s_Desc, FwdAlg, BwdDataAlg, BwdFilterAlg, InCh, P);




	//=======================================================================================================================================================
	//
	// The AllocateParamGPUMemory() function allocates the main GPU memory required to store the parameters of all weight layers, and their current 
	// derivatives, and the running averages of the squared derivatives of all trainable parameters. The definitions of these variables is provided in 
	// the Functions.cpp file that includes the implementation of the AllocateParamGPUMemory() function. The d_X and d_Y buffers used to store all 
	// input channels and output channels are allocated based on the maximum network input size. Also the d_F and the d_Indx buffers are also allocated
	// based on the maximum network input size.
	//
	// The AllocateAuxiliaryGPUMemory() function allocates additional GPU memory required to store some intermediate results, and store data required 
	// by few cuda kernels. The definitions of these additional variables is provided in the xxx.cu file that includes the implementation of the 
	// AllocateAuxiliaryGPUMemory() function. The d_YY buffer is allocated based on the maximum network input size.
	//
	//=======================================================================================================================================================	

	float **d_W, **d_V, **d_DW, **d_Ws, **d_Vs, **d_DWs, **d_X, **d_Y, **d_Param, **d_DParam, **d_ParamV, **d_SMU, **d_Derv;
	float *d_WF, *d_VF, *d_DWF, *d_YF, *d_Yv;
	int **d_Indx, WsSize[CL];
	bool **d_F;

	AllocateParamGPUMemory_Varm(&d_W, &d_V, &d_DW, &d_Ws, &d_Vs, &d_DWs, &d_X, &d_Y, &d_Param, &d_DParam, &d_ParamV, &d_SMU, &d_Derv, &d_WF, &d_VF, &d_DWF, &d_YF, &d_Yv, &d_Indx, &d_F, WSize, WsSize, P, Xc, Yc);

	float **d_SMUs, *d_YY, *d_Y0, *d_ws, *d_mse, *d_count, *d_rand1, *d_randRGB, *d_Cropf;
	int *Indx1, **Indx, *d_Indx1;
	unsigned int *d_Crop;

	AllocateAuxiliaryGPUMemory_Varm(&d_SMUs, &d_YY, &d_Y0, &d_ws, &d_rand1, &d_randRGB, &d_Cropf, &d_Crop, &d_mse, &d_count, &d_Indx1, &Indx, PStart, P, Yc, NParts);




	//=======================================================================================================================================================
	//
	// The InitializeCudaKernels() function initializes the following cuda kernel variables that define the thread structure of all cuda kernels used
	// in this program. In general there are the block size which defines the number and structure of the threads inside an individual theard block, 
	// and there is the grid size which defines the number and structure of these blocks of threads. Each of the theardsize or gridsize can be defined 
	// as a single integer variable or as a dim3 variable which is a cuda structure that contains 3 integer values. Most cuda kernels use similar block
	// sizes, and usually the grid size defines the structure of the cuda kernel. The following grid sizes define the thread structure of the main cuda 
	// kernels used in this program. Some cuda kernels have multiple sizes because the network is trained using a variable input size. The grid sizes for
	// such cuda kernels are initialized by declaring MunWin instants of the Var_gridSizes c struct.
	//
    // G:-                  a Var_gridSizes c struct that define cuda kernels with variables sizes.
	// G[k].gridSizeP[i] :- the grid size of the maxpooling kernel for conv layer i and for the kth input size.
	// G[k].gridSizeBN2[i] :- the grid size of the second stage of the forward pass and the first stage of the backward pass kernel of BN for conv layer i and for the kth input size.
	// G[k].gridSize_Crop :- the grid size for the data augmentation kernel for the kth input size.
	// G[k].gridSizeAddYB[i] :- the grid size used by the AddMatrix kernel which adds two matrices with equal sizes at layer i for the kth input size.
	//
	// gridSizeBN1[i] :-    the grid size for the first stage of the forward pass and the second stage of the backward pass kernel for BN of convolutional layer i.
	// gridSizeBN11[i] :-   the grid size for the first stage of the forward pass kernel for BN of convolutional layer i.
	// gridSizeAddA[i] :-   the grid size for the cuda kernel used to update the BN trainable parameters at layer i. 
	// gridSizeRGB :-       the grid size for the cuda kernel used to generate stochastic values used for colour augmentation.
	// gridSizePA :-        the grid size for the global average pooling used after the last convolutional layer. 
	// gridSizeAddWs[i] :-  the grid size for the cuda kernel used to update the weights of the residual connection at layer i. 
	// gridSizeAddW[i] :-   the grid size for the cuda kernel used to update the weights of convolutional layer i. 
	// gridSizeAddWF :-     the grid size for the cuda kernel used to update the weights of FC output layer.
	//
	//=======================================================================================================================================================	

	dim3  gridSizeBN1[CL], gridSizeBN11[CL], gridSizeAddA[CL], gridSizeRGB, gridSizePA;
	int gridSizeAddWs[CL], gridSizeAddW[CL], gridSizeAddWF;
	Var_gridSizes G[NumWin];

	InitializeCudaKernels_Var(G, gridSizeBN1, gridSizeBN11, gridSizeAddA, &gridSizeRGB, &gridSizePA, gridSizeAddWs, gridSizeAddW, &gridSizeAddWF, P, Yc, WSize, WsSize);




	//=======================================================================================================================================================
	// blockSize1 to 4 are thread block sizes defined to be used by the cuda kernels. alpha1 and alpha2 are used by cublas.lib and cudnn.lib functions.
	// wsZise is the work-space memory size required by the cudnn.lib convolution algorithms.
	//=======================================================================================================================================================	

	dim3 blockSize1(BLOCKSIZE1, 1, 1); dim3 blockSize2(BLOCKSIZE2, 1, 1); dim3 blockSize3(BLOCKSIZE3, 1, 1); dim3 blockSize4(BLOCKSIZE4, 1, 1);

	float alpha1 = 1.0f, beta1 = 0.0f, alpha2 = 1.0f, beta2 = 1.0f;
	uniform_int_distribution<int> distr_int(0, 1000000000);



	//=======================================================================================================================================================	
	// The Iter integer varaible is the training iteration count which will be used by RMSprop. The kk integer variable is used to reference the DecAlpha[] array
	// which is used to define the intervals where the training rate will be decayed.
	//
	//
	// The ParameterInitialization() function initializes the trainable parameters of the network. It initializes d_W the weights of all conv layers, 
	// d_WF the weights of the FC output layer, and d_Ws the weights of the conv layers of the residual connections using Kamming initialization.
	// It also initializes d_Param the trainable parameters of the batch normalization layers.
	//=======================================================================================================================================================	

	int i0 = 0, kk0 = 0, Iter0 = 0;

	//=======================================================================================================================================================	
	// If training starts from random weights then the ParameterInitialization() function is used to initialize the network weights using Kaiming initialization.
	//=======================================================================================================================================================	

	if (Operation_Mode == TRAIN || Operation_Mode == TRAIN_PLUS_INFERENCE)
	{
		ParameterInitialization(d_W, d_Ws, d_WF, d_Param, TEMP, WSize, WsSize);
	}

	//=======================================================================================================================================================	
	// When carrying inference only, the network parameters are initialized by loading the parameters of a previously trained network using the inParam
	// file pointer.
	//=======================================================================================================================================================	

	if (Operation_Mode == INFERENCE)
	{
		ReloadParameters1(inParam, TEMP, d_W, d_Ws, d_WF, d_Param, WSize, WsSize, In1);
	}

	//=======================================================================================================================================================	
	// If training was interuppted, then the network parameters (including the running averages of the derivatives) are initailzed by loading the last 
	// saved copy before interupption happened.
	//=======================================================================================================================================================	

	if (Operation_Mode == INTERRUPTED_TRAIN || Operation_Mode == INTERRUPTED_TRAIN_PLUS_INFERENCE)
	{
		ReloadParameters2(inParam, TEMP, d_W, d_V, d_Ws, d_Vs, d_WF, d_VF, d_Param, d_ParamV, WSize, WsSize, In1, &i0, &Iter0, &kk0, &lr);
	}

	int Iter = Iter0, kk = kk0;

	//=======================================================================================================================================================	
	//
	//                                    Start of the training phase
	// Two CPU threads are executed in a producer-consumer relationship, where the ReadFile reads the input images from the disk drive to the main memeory
	// and the main CPU threads consumes the data stored by the ReadFile thread, and use it to train the network. We find that for the 34 Layer residual 
	// network such implementation elementates the latency of reading from a mid range 500 MB/s sata SSD drive.
	//=======================================================================================================================================================	


	if (Operation_Mode == TRAIN || Operation_Mode == INTERRUPTED_TRAIN || Operation_Mode == TRAIN_PLUS_INFERENCE || Operation_Mode == INTERRUPTED_TRAIN_PLUS_INFERENCE)
	{
		//=======================================================================================================================================================	
		// ReadFile is the second CPU thread that plays the producer part in the producer-consumer relationship with the main thread (program). The ReadFile
		// thread reads the input channels into one half of the 3 RGB buffers, and the main program consumes the avaliable data in the other half. 
		//=======================================================================================================================================================	

		thread t1(ReadFile, Red, Green, Blue, PartSize, NParts, VParts, i0);
		this_thread::sleep_for(chrono::seconds(10));

		//---------------------------------------------------------------------------------------------------------

		cudaStatus = hipEventRecord(start, 0);

		//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
		//
		//                                                           Main Forloop main CPU thread
		//
		// This the main training for loop where CUDA kernels and functions are called to train a deep residual CNN on an NVIDIA GPU. The first part of the 
		// for loop implements the forward pass, and the second part implements the backward pass, the backpropagation of the error signal signal to update
		// the trainable parameters of all layers. Because the training+validation input RGB channels are divided into (NParts) segments to fit into main 
		// memory this for loop is executed Epoch*NParts times where Epoch is the number of the training epoches.
		//
		//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%

		for (int i = i0; (i < NParts*NumEpoch) && MSE < 100; i++)
		{
			//------------------------------------------------------------------------------------------------------------------------------------------------------	
			// p:- is the current data segment index, epoch:- is the current epoch index, alter:- makes the main CPU thread alternate between the two halfs of the 
			// of the GPU buffers that contain the RGB input channels (producer-consumer relationship).
			//------------------------------------------------------------------------------------------------------------------------------------------------------	

			int p = i%NParts;
			int epoch = i / NParts;
			size_t altr = (i % 2)*(RGB_GPU_SIZE / 2);

			//------------------------------------------------------------------------------------------------------------------------------------------------------	
			// The hiprandGenerate() hiprand.lib function generates 3 integer random values per image, 2 used to decide the cropping postion and 1 to decide on
			// horizontal flipping. hiprandGenerateUniform() hiprand.lib function generates 2 uinform random values between 0 and 1 for each images, one used to 
			// decide the amount of scaling, and the other used to decide on the amount of aspect ratio augmenation. The hiprandGenerateNormal() generates 3 random
			// variables per image to be used by the RGBrandPCA() cuda kernel to generate 3 stochastic numbers to be added to the 3 RGB input channels for 
			// colour augmentation.
			//------------------------------------------------------------------------------------------------------------------------------------------------------	

			if (p == 0)
			{
				hiprandStatus = hiprandGenerate(cuda_gen, d_Crop, 3 * TrainSizeM);
				hiprandStatus = hiprandGenerateUniform(cuda_gen, d_Cropf, 2 * TrainSizeM);
				hiprandStatus = hiprandGenerateNormal(cuda_gen, d_rand1, 3 * TrainSizeM, 0.0f, 0.1f);
				RGBrandPCA<TrainSizeM> << <gridSizeRGB, blockSize1 >> >(d_randRGB, d_rand1);
			}

			//-----------------------------------------------------------------------------------------------------------------------------------------	
			// The ReshuffleImages() function randomly reshuffles the input images within a single data segment.
			//-----------------------------------------------------------------------------------------------------------------------------------------	

			Indx1 = Indx[p];
			ReshuffleImages(Indx1, PStart, p);
			cudaStatus = hipMemcpyAsync(d_Indx1, Indx1, sizeof(int) * (PStart[p + 1] - PStart[p]), hipMemcpyHostToDevice);


			cudaStatus = hipMemsetAsync(d_count, 0, sizeof(float));
			cudaStatus = hipMemsetAsync(d_mse, 0, sizeof(float)*BatchSize);

			//=========================================================================================================================================	

			for (int j = PStart[p]; j < PStart[p + 1]; j += BatchSize)
			{
				//-----------------------------------------------------------------------------------------------------------------------------------------	
				// At the start of each iteration the DataAugmentation() CUDA kernel is called to generate a batch after applying data augmentaion. For the 
				// training data-segements (p < NParts - VParts) DataAugmentation() is used, and for the validation data-segments (NParts - VParts) >= p >
				// NParts DataAugmentationValidate() is used.
				//
				// The integer variable o is the index of the randomly chosen input variable. For the training dataset (p < NParts - VParts) the input size
				// is randomly chosen from the set of predefined input sizes (o = distr_int(gen) % NumWin;) except for the last iteration (epoch = NumEpoch-1;)
				// where the median input size is used (o = NumWin/2;) to generate the fixed means and variances of BN that will be used in the inference stage.
				// For the validation set the median input size is used for the single crop validation to measure the performance of the network during training.
				// For the selected input size o, the corresponding grid size (G[o].gridSize_Crop) is used for the data augmentation cuda kernels.
				//-----------------------------------------------------------------------------------------------------------------------------------------	

				int o;

				if (p < NParts - VParts)
				{
					Iter++;
					o = (epoch<(NumEpoch - 1)) ? distr_int(gen) % NumWin : NumWin / 2;
					DataAugmentation <1> << <G[o].gridSize_Crop, blockSize1 >> >(d_X[0], Red + altr, Green + altr, Blue + altr, d_HeightTr + PStart[p], d_WidthTr + PStart[p], d_StartTr + PStart[p], d_Indx1 + j - PStart[p], (d_Crop + 3 * j), (d_randRGB + 3 * j), (d_Cropf + 2 * j), P[o].IR[0]);
				}
				else
				{
					o = NumWin / 2;
					DataAugmentationValidate <1> << <G[o].gridSize_Crop, blockSize1 >> >(d_X[0], Red + altr, Green + altr, Blue + altr, d_HeightTr + PStart[p], d_WidthTr + PStart[p], d_StartTr + PStart[p], d_Indx1 + j - PStart[p], P[o].IR[0]);
				}

				//=========================================================================================================================================	

				//-----------------------------------------------------------------------------------------------------------------------------------------	
				// Once the training batch was generated this batch will be passed forward through all CL convolutional layers. Therefore the next
				// for loop which will be repeated CL times will propagate the input signal froward through all CL convolutional layers. At each 
				// layer, the hipdnnConvolutionForward() cudnn.lib function implements the forward pass of the convolution operation, and then the 
				// BatchNormForward1x() and BatchNormForward2() cuda kernels implements the forward pass of the BN operation. For layer k with 
				// PoolType[k] =1 the MaxPoolingForward() cuda kernel implements the forward pass of the max pooling operation. For layer k with 
				// PoolType[k] = 2 the GlobalAvgPoolingForward() cuda kernel implements the forward pass of the global average pooling operation.    
				// The variable JMP defines the number of convolutional layer that will be skipped by a residual connection. The condition 
				// (k > 0 && k%JMP == 0) means that layer k has an additional residual input coming from layer k-JMP. For such layers BatchNormForward22() 
				// will be used instead of BatchNormForward2() because the residual input is incorporated after BN and before ReLU activation, and both 
				// are combined in one cuda kernel to reduce GPU memory loads/stores.
				//
				// The input and output tensor descriptors for the chosen input size o (Desc_X[o][k] and Desc_Y[o][k]) are used with the hipdnnConvolutionForward() 
				// cudnn.lib function that implements the forward pass of the conv layer k.
				//
				// The input and output tensor descriptors for the chosen input size o (Desc_Xs[o][k] and Desc_Ys[o][k]) are used with the hipdnnConvolutionForward() 
				// cudnn.lib function that implements the forward pass of the conv layer of the residual connection that passes the output of layer k-JMP to layer k.
				//
				// For the chosen input size o, the corresponding gird size (G[o].gridSizeBN2[k]) is used with the BatchNormForward2 and BatchNormForward22
				// cuda kernels that implements the second stage of the forward pass of BN layer k.
				//
				// For the chosen input size o, the corresponding gird size (G[o].gridSizeP[k]) is used with the MaxPoolingForward cuda kernel that implements
				// the forward pass of maxpooling after conv layer k.
				//
				// For the chosen input size o, the corresponding conv and pooling parameters (P[o].CR[k], P[o].SR[k], and P[o].Yr[k]) are used with the 
				// various cuda kernel. Again o is the index of the chosen input size, while  WinSize[o] (or P[o].IR[0]) is the size itself.
				//-----------------------------------------------------------------------------------------------------------------------------------------	

				float *d_ts, *d_t;

				for (int k = 0; k < CL; k++)
				{
					//-----------------------------------------------------------------------------------------------------------------------------------------	

					dnnStatus = hipdnnConvolutionForward(dnnHandle, &alpha1, Desc_X[o][k], d_X[k], Desc_W[k], d_W[k], Conv_Desc[k], FwdAlg[k], d_ws, wsSize, &beta1, Desc_Y[o][k], d_Y[k]);

					//-----------------------------------------------------------------------------------------------------------------------------------------	

					BatchNormForward1a <BLOCKSIZE4> << <gridSizeBN1[k], blockSize4 >> >(d_SMU[k], d_Y[k], OutCh[k], P[o].Yr[k]);
					if (epoch == (NumEpoch - 1) && p < (NParts - VParts))  BatchNormForwardT1b<BLOCKSIZE4> << <gridSizeBN11[k], blockSize4 >> >(d_SMU[k], d_SMUs[k], OutCh[k], P[o].Yr[k] * BatchSize);
    				else                                                   BatchNormForward1b<BLOCKSIZE4> << <gridSizeBN11[k], blockSize4 >> >(d_SMU[k], OutCh[k], P[o].Yr[k] * BatchSize);

					//-----------------------------------------------------------------------------------------------------------------------------------------	

					d_ts = (k < CL - 1) ? d_X[k + 1] : d_Y0;


					if (k > 0 && k%JMP == 0)
					{
						d_t = d_X[k - JMP + 1];
						if (OutCh[k - JMP] != OutCh[k])
						{
							dnnStatus = hipdnnConvolutionForward(dnnHandle, &alpha1, Desc_Xs[o][k - JMP], d_X[k - JMP + 1], Desc_Ws[k - JMP], d_Ws[k - JMP], Conv_s_Desc, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM, d_ws, wsSize, &beta1, Desc_Ys[o][k - JMP], d_YY);
							d_t = d_YY;
						}

						if (PoolType[k] == 0)
						{
							BatchNormForward22 <BLOCKSIZE4> << <G[o].gridSizeBN2[k], blockSize4 >> >(d_ts, d_Y[k], d_t, d_F[k], d_SMU[k], d_Param[k], OutCh[k], P[o].Yr[k]);
						}

						if (PoolType[k] == 1)
						{
							BatchNormForward22 <BLOCKSIZE4> << <G[o].gridSizeBN2[k], blockSize4 >> >(d_YY, d_Y[k], d_t, d_F[k], d_SMU[k], d_Param[k], OutCh[k], P[o].Yr[k]);
							MaxPoolingForward <1> << <G[o].gridSizeP[k], blockSize1 >> >(d_ts, d_YY, d_Indx[k], P[o].CR[k], P[o].SR[k], Sr1[k], Sr2[k], OutCh[k]);
						}

						if (PoolType[k] == 2)
						{
							BatchNormForward22 <BLOCKSIZE4> << <G[o].gridSizeBN2[k], blockSize4 >> >(d_YY, d_Y[k], d_t, d_F[k], d_SMU[k], d_Param[k], OutCh[k], P[o].Yr[k]);
							GlobalAvgPoolingForward <BLOCKSIZE3> << <gridSizePA, blockSize3 >> >(d_ts, d_YY, OutCh[k], P[o].CR[k] * P[o].CR[k]);
						}

					}
					else
					{
						if (PoolType[k] == 0)
						{
							BatchNormForward2 <BLOCKSIZE4> << <G[o].gridSizeBN2[k], blockSize4 >> >(d_ts, d_Y[k], d_SMU[k], d_Param[k], OutCh[k], P[o].Yr[k]);
						}

						if (PoolType[k] == 1)
						{
							BatchNormForward2 <BLOCKSIZE4> << <G[o].gridSizeBN2[k], blockSize4 >> >(d_YY, d_Y[k], d_SMU[k], d_Param[k], OutCh[k], P[o].Yr[k]);
							MaxPoolingForward <1> << <G[o].gridSizeP[k], blockSize1 >> >(d_ts, d_YY, d_Indx[k], P[o].CR[k], P[o].SR[k], Sr1[k], Sr2[k], OutCh[k]);
						}

						if (PoolType[k] == 2)
						{
							BatchNormForward2 <BLOCKSIZE4> << <G[o].gridSizeBN2[k], blockSize4 >> >(d_YY, d_Y[k], d_SMU[k], d_Param[k], OutCh[k], P[o].Yr[k]);
							GlobalAvgPoolingForward <BLOCKSIZE3> << <gridSizePA, blockSize3 >> >(d_ts, d_YY, OutCh[k], P[o].CR[k] * P[o].CR[k]);
						}
					}
				}

				//-----------------------------------------------------------------------------------------------------------------------------------------	
				// the hipblasSgemm() cuda matrix multiplication function implements the forward pass of the FC output layer. The SoftmaxForward() 
				// cuda kernel applies softmax activation to the outputs of the hipblasSgemm() function and returns the error signal propagated back 
				// through the softmax stage in d_YF.
				//-----------------------------------------------------------------------------------------------------------------------------------------	

				blasstatus = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, Out1, BatchSize, In1 + 1, &alpha1, d_WF, Out1, d_Y0, In1 + 1, &beta1, d_YF, Out1);

				Softmax< NumClasses, BLOCKSIZE2> << <BatchSize, BLOCKSIZE2 >> >(d_YF, d_T + PStart[p], d_Indx1 + j - PStart[p], d_mse, d_count);
			

				//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
				//                                                       Back-Propagation                                                                || 
				//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%


				//-----------------------------------------------------------------------------------------------------------------------------------------	
				// NParts is the total number of data-segments which include the training and validation data-segments, and VParts is the number of 
				// validation data-segments only. p is the index of the current data-segment. The backward pass should only be executed for the training 
				// data-segments, and not for the validation data-segments, and therefore should only be executed for p < (NParts - VParts). The last 
				// epoch in this implementation is dedicated to calculate the fixed means and variances of BN that will be used in the inference stage,
				// which only requires the forward pass, and therefore the backward pass should only be executed for epoch < (Epoch - 1).
				//-----------------------------------------------------------------------------------------------------------------------------------------	


				if (p < (NParts - VParts) && epoch < (NumEpoch - 1))
				{
					//-----------------------------------------------------------------------------------------------------------------------------------------	
					// The first stage in the back-propagation half is propagating the error signal back through the FC output layer. The first hipblasSgemm() 
					// function calculates the derivatives of the weights of the FC output layer in d_DWF. The second hipblasSgemm() function propagates back 
					// the error signal from the outputs of the output layer d_YF to the inputs of the output layer d_Y0.
					//-----------------------------------------------------------------------------------------------------------------------------------------	

					blasstatus = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, Out1, In1 + 1, BatchSize, &alpha1, d_YF, Out1, d_Y0, In1 + 1, &beta1, d_DWF, Out1);

					blasstatus = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, In1, BatchSize, Out1, &alpha1, d_WF, Out1, d_YF, Out1, &beta1, d_Y0, In1 + 1);

					//-----------------------------------------------------------------------------------------------------------------------------------------
					// Then the error signal is propagated back through all CL convolutional layers to update all trainable parameters in those layers. 
					// d_DParam[k] and d_Derv[k] are updated by accumulating different thread block results using atomicadds() and therefore they 
					// need to be reset to zero. The error signal propagates back through a convolutional layer in a reverse order, through the pooling
					// stage (if there is one), then through the BN stage, and finally through the convolution stage. 
					// For conv layer k where (k%JMP == 0 && k + JMP < CL) two backpropagated signals will be propagated back to that layer's 
					// output, one from conv layer k+1 and one from the residual connection that connects this output forward to layer k+JMP.
					// For conv layer = where (k > 0 && k%JMP == 0) the BatchNormBackward22() BN kernel will be used instead of BatchNormBackward2(),
					// because that layer has an additional residual input coming from layer k-JMP.
					// The BatchNormBackward2() and BatchNormBackward1() kernels implement the backward pass for BN.
					// For (PoolType[k] == 1) the MaxPoolingBackward() kernel implements the backward pass for the maxpooling stage for conv layer k.
					// For (PoolType[k] == 2) the GlobalAvgPoolingBackward() kernel implements the backward pass for the average pooling stage for conv layer k.
					// The hipdnnConvolutionBackwardFilter(k) cudnn.lib function calculates the derivatives for the weights of conv layer k.
					// The hipdnnConvolutionBackwardData(k) cudnn.lib function propagates the error signal back from the output channels side to the input
					// channels side for conv layer k.
					//
					// The input and output tensor descriptors for the chosen input size o (Desc_X[o][k] and Desc_Y[o][k]) are used with the hipdnnConvolutionBackwardFilter() 
					// cudnn.lib function that calculates the derivatives used to update the weights of conv layer k, and also used with the hipdnnConvolutionBackwardData()
					// cudnn.lib function which implements the backward pass for conv layer k.
					//
					// The input and outpout tensor descriptors for the chosen input size o (Desc_Xs[o][k] and Desc_Ys[o][k]) are used with the hipdnnConvolutionBackwardFilter() 
					// and hipdnnConvolutionBackwardData() cudnn.lib functions that calculates the derivatives and implements the backward pass of the conv layer of the residual 
					// connection that passes the output of layer k to layer k+JMP.
					//
					// For the chosen input size o, the corresponding gird size (G[o].gridSizeBN2[k]) is used with the BatchNormBackward1 cuda kernel that implements 
					// the second stage of the backward pass of BN layer k.
					//
					// For the chosen input size o, the corresponding gird size (G[o].gridSizeP[k]) is used with the MaxPoolingBackward cuda kernel that implements
					// the backward pass of maxpooling after conv layer k.
					//
					// For the chosen input size o, the corresponding conv and pooling parameters (P[o].CR[k], P[o].SR[k], P[o].Xr[k], and P[o].Yr[k]) are used with  
					// various cuda kernel.

					//-----------------------------------------------------------------------------------------------------------------------------------------

					for (int k = CL - 1; k >= 0; k--)
					{
						cudaStatus = hipMemsetAsync(d_DParam[k], 0, sizeof(float) * 2 * OutCh[k]);
						cudaStatus = hipMemsetAsync(d_Derv[k], 0, sizeof(float) * 2 * OutCh[k]);

						//-----------------------------------------------------------------------------------------------------------------------------------------					

						d_ts = (k < CL - 1) ? d_X[k + 1] : d_Y0;

						if (k%JMP == 0 && k + JMP < CL)
						{
							d_t = (PoolType[k + JMP] == 0) ? d_X[k + JMP + 1] : d_YY;
							if (OutCh[k] == OutCh[k + JMP])   Add_Mtx <1> << <G[o].gridSizeAddYB[k], BLOCKSIZE1 >> >(d_ts, d_t, P[o].Xr[k + 1] * Xc[k + 1]);
							else							  dnnStatus = hipdnnConvolutionBackwardData(dnnHandle, &alpha2, Desc_Ws[k], d_Ws[k], Desc_Ys[o][k], d_t, Conv_s_Desc, HIPDNN_CONVOLUTION_BWD_DATA_ALGO_1, d_ws, wsSize, &beta2, Desc_Xs[o][k], d_ts);
						}

						//-----------------------------------------------------------------------------------------------------------------------------------------
						if (k > 0 && k%JMP == 0)
						{
							if (PoolType[k] == 0)
							{
								BatchNormBackward22 <BLOCKSIZE4> << <gridSizeBN1[k], blockSize4 >> >(d_DParam[k], d_Derv[k], d_Param[k], d_SMU[k], d_ts, d_F[k], d_Y[k], OutCh[k], P[o].Yr[k]);
								BatchNormBackward1 <BLOCKSIZE4> << <G[o].gridSizeBN2[k], blockSize4 >> >(d_Y[k], d_ts, d_Param[k], d_SMU[k], d_Derv[k], OutCh[k], P[o].Yr[k]);
							}

							if (PoolType[k] == 1)
							{
								cudaStatus = hipMemsetAsync(d_YY, 0, P[o].Yr[k] * Yc[k] * sizeof(float));
								MaxPoolingBackward <1> << <G[o].gridSizeP[k], blockSize1 >> >(d_YY, d_ts, d_Indx[k], P[o].SR[k] * P[o].SR[k], OutCh[k]);
								BatchNormBackward22 <BLOCKSIZE4> << <gridSizeBN1[k], blockSize4 >> >(d_DParam[k], d_Derv[k], d_Param[k], d_SMU[k], d_YY, d_F[k], d_Y[k], OutCh[k], P[o].Yr[k]);
								BatchNormBackward1 <BLOCKSIZE4> << <G[o].gridSizeBN2[k], blockSize4 >> >(d_Y[k], d_YY, d_Param[k], d_SMU[k], d_Derv[k], OutCh[k], P[o].Yr[k]);
							}

							if (PoolType[k] == 2)
							{
								GlobalAvgPoolingBackward <BLOCKSIZE3> << <gridSizePA, blockSize3 >> >(d_YY, d_ts, OutCh[k], P[o].CR[k] * P[o].CR[k]);
								BatchNormBackward22 <BLOCKSIZE4> << <gridSizeBN1[k], blockSize4 >> >(d_DParam[k], d_Derv[k], d_Param[k], d_SMU[k], d_YY, d_F[k], d_Y[k], OutCh[k], P[o].Yr[k]);
								BatchNormBackward1 <BLOCKSIZE4> << <G[o].gridSizeBN2[k], blockSize4 >> >(d_Y[k], d_YY, d_Param[k], d_SMU[k], d_Derv[k], OutCh[k], P[o].Yr[k]);
							}
						}
						else
						{
							if (OutCh[k - 1] != OutCh[k + JMP - 1] && (k + JMP - 1) % JMP == 0)
							{
								d_t = (PoolType[k + JMP - 1] == 0) ? d_X[k + JMP] : d_YY;
								dnnStatus = hipdnnConvolutionBackwardFilter(dnnHandle, &alpha1, Desc_Xs[o][k - 1], d_X[k], Desc_Ys[o][k - 1], d_t, Conv_s_Desc, HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_1, d_ws, wsSize, &beta1, Desc_Ws[k - 1], d_DWs[k - 1]);
							}

							if (PoolType[k] == 0)
							{
								BatchNormBackward2 <BLOCKSIZE4> << <gridSizeBN1[k], blockSize4 >> >(d_DParam[k], d_Derv[k], d_Param[k], d_SMU[k], d_ts, d_Y[k], OutCh[k], P[o].Yr[k]);
								BatchNormBackward1 <BLOCKSIZE4> << <G[o].gridSizeBN2[k], blockSize4 >> >(d_Y[k], d_ts, d_Param[k], d_SMU[k], d_Derv[k], OutCh[k], P[o].Yr[k]);
							}

							if (PoolType[k] == 1)
							{

								cudaStatus = hipMemsetAsync(d_YY, 0, P[o].Yr[k] * Yc[k] * sizeof(float));
								MaxPoolingBackward <1> << <G[o].gridSizeP[k], blockSize1 >> >(d_YY, d_ts, d_Indx[k], P[o].SR[k] * P[o].SR[k], OutCh[k]);
								BatchNormBackward2 <BLOCKSIZE4> << <gridSizeBN1[k], blockSize4 >> >(d_DParam[k], d_Derv[k], d_Param[k], d_SMU[k], d_YY, d_Y[k], OutCh[k], P[o].Yr[k]);
								BatchNormBackward1 <BLOCKSIZE4> << <G[o].gridSizeBN2[k], blockSize4 >> >(d_Y[k], d_YY, d_Param[k], d_SMU[k], d_Derv[k], OutCh[k], P[o].Yr[k]);
							}

							if (PoolType[k] == 2)
							{
								GlobalAvgPoolingBackward <BLOCKSIZE3> << <gridSizePA, blockSize3 >> >(d_YY, d_ts, OutCh[k], P[o].CR[k] * P[o].CR[k]);
								BatchNormBackward2 <BLOCKSIZE4> << <gridSizeBN1[k], blockSize4 >> >(d_DParam[k], d_Derv[k], d_Param[k], d_SMU[k], d_YY, d_Y[k], OutCh[k], P[o].Yr[k]);
								BatchNormBackward1 <BLOCKSIZE4> << <G[o].gridSizeBN2[k], blockSize4 >> >(d_Y[k], d_YY, d_Param[k], d_SMU[k], d_Derv[k], OutCh[k], P[o].Yr[k]);
							}
						}

						//-----------------------------------------------------------------------------------------------------------------------------------------

						dnnStatus = hipdnnConvolutionBackwardFilter(dnnHandle, &alpha1, Desc_X[o][k], d_X[k], Desc_Y[o][k], d_Y[k], Conv_Desc[k], BwdFilterAlg[k], d_ws, wsSize, &beta1, Desc_W[k], d_DW[k]);

						//-----------------------------------------------------------------------------------------------------------------------------------------

						if (k > 0) dnnStatus = hipdnnConvolutionBackwardData(dnnHandle, &alpha1, Desc_W[k], d_W[k], Desc_Y[o][k], d_Y[k], Conv_Desc[k], BwdDataAlg[k], d_ws, wsSize, &beta1, Desc_X[o][k], d_X[k]);

						//-----------------------------------------------------------------------------------------------------------------------------------------

					}

					//-----------------------------------------------------------------------------------------------------------------------------------------
					// Once all the derivatives are ready the  Update_RMSprop1() cuda kernel is invoked to update the weights of the conv layers, and the 
					// weights of the residual connections, and the weights of the FC output layer. The Update_RMSprop2() is called to update the trainable
					// parameters of the BN stages. Update_RMSprop1() and Update_RMSprop2() use RMSprop version of SGD to update the network parameters.  
					//-----------------------------------------------------------------------------------------------------------------------------------------

					for (int k = 0; k < CL; k++)
					{
						Update_RMSprop1<1> << <gridSizeAddW[k], BLOCKSIZE1 >> >(d_W[k], d_V[k], d_DW[k], lr, lmda, WSize[k], Iter);
						Update_RMSprop2<1> << <gridSizeAddA[k], blockSize1 >> >(d_Param[k], d_ParamV[k], d_DParam[k], lr, lmda, 2 * OutCh[k], Iter);
					}

					for (int k = 0; k < CL - JMP; k += JMP)
					{
						if (OutCh[k] != OutCh[k + JMP])
							Update_RMSprop1<1> << <gridSizeAddWs[k], BLOCKSIZE1 >> >(d_Ws[k], d_Vs[k], d_DWs[k], lr, lmda, WsSize[k], Iter);
					}

					Update_RMSprop1<1> << <gridSizeAddWF, BLOCKSIZE1 >> >(d_WF, d_VF, d_DWF, lr, lmda, Out1*(In1 + 1), Iter);

				}

			}

			//-----------------------------------------------------------------------------------------------------------------------------------------
			// PrintIterResults() prints the mse and classification rates for the training and validation sets per epoch. Then the learning rate 
			// is decayed at specific intervals (specific epochs) defined in array DecAlpha[]. SaveParameters2() saves the network parameters 
			// including the running averages required by RMSprop once in every 10 epochs in case training was interrupted the network can resume
			// training from the last saved set of parameters. 
			//-----------------------------------------------------------------------------------------------------------------------------------------

			PrintIterResults(outResults, d_mse, d_count, PStart, NParts, VParts, i);

			if (p == (NParts - 1) && (epoch + 1) == DecAlpha[kk]) { lr = lr*0.4f; kk++; }

			if ((epoch + 1) % 5 == 0) SaveParameters2(outParamCopy, TEMP, d_W, d_V, d_Ws, d_Vs, d_WF, d_VF, d_Param, d_ParamV, WSize, WsSize, In1, i + 1, Iter, kk, lr);

			//-----------------------------------------------------------------------------------------------------------------------------------------
			// One the current training epoch is finished the main CPU thread synchronizes with the other thread that reads the data from the desk
			// to the main memory (producer-consumer synchronization). This guarantees that there is data available to be consumed by the main thread 
			// that runs the CNN.
			//-----------------------------------------------------------------------------------------------------------------------------------------

			unique_lock<mutex> locker(mu);

			slot--;
			while (slot == 0 && i < (NParts*NumEpoch - 1) && MSE < 100)
				not_empty.wait(locker);

			not_full.notify_one();
			locker.unlock();

		}

		//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
		//
		//   End of main Forloop, end of training phase
		//
		//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%

		t1.join();

		SaveParameters1(outParam, TEMP, d_W, d_Ws, d_WF, d_Param, WSize, WsSize, In1);
		fclose(outParam);
		fclose(outParamCopy);



		//=========================================================================================================================================
		// AdjustFixedMeansStds() averages the accumulated means and varainces in d_SMUs to be used as a fixed set in the inference stage.
		//=========================================================================================================================================

		for (int i = 0; i < CL; i++)
		{
			AdjustFixedMeansStds <1> << <gridSizeAddA[i], blockSize1 >> >(d_SMUs[i], 2 * OutCh[i]);
			cudaStatus = hipMemcpy(TEMP, d_SMUs[i], sizeof(float) * 2 * OutCh[i], hipMemcpyDeviceToHost);
			fwrite(TEMP, sizeof(float), 2 * OutCh[i], outMeansVariances);
		}
		fclose(outMeansVariances);
	}




	//=========================================================================================================================================
	// hipEventElapsedTime() runtime cuda function returns the total time in milliseconds spent in the training phase. SaveParameters() saves 
	// the network parameters. FreeTrainSpecificData() frees all memory that is required in the training stage but it is not required in the 
	// inference stage.
	//=========================================================================================================================================

	cudaStatus = hipEventRecord(stop, 0);
	cudaStatus = hipEventSynchronize(stop);
	float time1;
	cudaStatus = hipEventElapsedTime(&time1, start, stop);

	cout << endl << "time = " << time1 << endl << endl;
	fprintf(outResults, "\n\ntime = %f\n\n ", time1);

	FreeTrainSpecificData(d_V, d_DW, d_Vs, d_DWs, d_DParam, d_ParamV, d_Derv, d_VF, d_DWF, d_count, d_rand1, d_randRGB, d_Cropf, d_Indx1, Indx, Indx1, d_Crop, NParts);


	//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
	//
	//                                                       Inference phase.
	//
	//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%

	if (Operation_Mode == INFERENCE || Operation_Mode == TRAIN_PLUS_INFERENCE || Operation_Mode == INTERRUPTED_TRAIN_PLUS_INFERENCE)
	{
		//======================================================================================================================================================
		// In the case of doing inference only, a previously stored copy of the fixed means and variances (produced by the same network used for inference) are 
		// reloaded into d_SMUs (using the inMeansVariances file pointer).  
		//======================================================================================================================================================

		if (Operation_Mode == INFERENCE)
		{
			for (int i = 0; i < CL; i++)
			{
				size_t numread = fread(TEMP, sizeof(float), 2 * OutCh[i], inMeansVariances);
				cudaStatus = hipMemcpy(d_SMUs[i], TEMP, sizeof(float) * 2 * OutCh[i], hipMemcpyHostToDevice);
			}
			fclose(inMeansVariances);
		}

		int *d_MTX;

		//======================================================================================================================================================
		// The InitializeTrainingData() function initializes the d_HeightTr, d_WidthTr, and d_StartTr GPU buffers with the heights, widths, and starting 
		// memory addresses of the test images. InitializeMultiCropInference() allocates and initializes the d_MTX GPU buffer with the cropping positions and 
		// and scales that will be used with multi-crop Inference. The three input files in6, in7, and in8 are opened to read the RGB input channels of the 
		// test images.
		//======================================================================================================================================================

		InitializeTrainingData(&d_HeightTr, &d_WidthTr, &d_StartTr, &d_T, &PStart, &PartSize, &StartPart, &NParts, &VParts, TestSizeM);
		InitializeMultiCropInference(&d_MTX);
		FILE *in6, *in7, *in8;

		char FileName[128];

		strcpy(FileName, DataFloder); strcat(FileName, "valRed.txt");
		in6 = fopen(FileName, "rb");
		strcpy(FileName, DataFloder); strcat(FileName, "valGreen.txt");
		in7 = fopen(FileName, "rb");
		strcpy(FileName, DataFloder); strcat(FileName, "valBlue.txt");
		in8 = fopen(FileName, "rb");



		//=====================================================================================================================================================================	
		// Alocate d_Yss GPU buffer that will be used to store the whole output labels predicted by the network. Alocate d_flip GPU buffer that will be initialized by 
		// random integers used to decide on horizontal flipping.
		//=====================================================================================================================================================================	

		float  *d_Yss;
		cudaStatus = hipMalloc(&d_Yss, sizeof(float) * NumClasses*TestSize);
		cudaStatus = hipMemset(d_mse, 0, sizeof(float)*BatchSize);
		cudaStatus = hipMemset(d_Yss, 0, sizeof(float)*TestSizeM*NumClasses);

		unsigned int *d_flip;
		cudaStatus = hipMalloc(&d_flip, sizeof(int)*TestSizeM*EpochTs);




		//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
		//
		//                                                      Start of the inference Forloop.
		//
		//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%

		for (int i = 0; i < NParts; i++)
		{
			//-----------------------------------------------------------------------------------------------------------------------------------------
			// Even though training was carried out using a variable input size, the inference is carried out using one network input size which is the
			// median size o = NumWin / 2; This simplifies and speeds up the inference stage.
			//-----------------------------------------------------------------------------------------------------------------------------------------

			int p = i%NParts;
			int o = NumWin / 2;
			//=========================================================================================================================================	
			// hiprandGenerate() initializes d_flip with random integer numbers. The fread() function reads one data segment of the test set input RGB 
			// channels into the Red, Green, and Blue buffers.
			//=========================================================================================================================================	

			hiprandStatus = hiprandGenerate(cuda_gen, d_flip, (PStart[p + 1] - PStart[p])*EpochTs);

			size_t numread10 = fread(Red, sizeof(unsigned char), PartSize[p], in6);
			size_t numread11 = fread(Green, sizeof(unsigned char), PartSize[p], in7);
			size_t numread12 = fread(Blue, sizeof(unsigned char), PartSize[p], in8);

			//=========================================================================================================================================	
			// Once the test data is ready in the memory, the test images are passed forward to calculate the test image labels in d_Yss. 
			//=========================================================================================================================================	

			for (int j = PStart[p]; j < PStart[p + 1]; j += BatchSize)
			{
				if (j % 500 == 0)cout << j << "   ";

				for (int epoch = 0; epoch < EpochTs; epoch++)
				{
					int indxf = BatchSize*epoch + EpochTs*(j - PStart[p]);
					DataAugmentationInference<EpochTs> << <G[o].gridSize_Crop, blockSize1 >> >(d_X[0], Red, Green, Blue, d_HeightTr + j, d_WidthTr + j, d_StartTr + j, d_MTX + j * 3 * EpochTs, d_flip + indxf, epoch, P[o].IR[0]);

					//=========================================================================================================================================	

					float *d_ts, *d_t;

					for (int k = 0; k < CL; k++)
					{
						//-----------------------------------------------------------------------------------------------------------------------------------------	

						dnnStatus = hipdnnConvolutionForward(dnnHandle, &alpha1, Desc_X[o][k], d_X[k], Desc_W[k], d_W[k], Conv_Desc[k], FwdAlg[k], d_ws, wsSize, &beta1, Desc_Y[o][k], d_Y[k]);

						//-----------------------------------------------------------------------------------------------------------------------------------------	

						d_ts = (k < CL - 1) ? d_X[k + 1] : d_Y0;


						if (k > 0 && k%JMP == 0)
						{
							d_t = d_X[k - JMP + 1];
							if (OutCh[k - JMP] != OutCh[k])
							{
								dnnStatus = hipdnnConvolutionForward(dnnHandle, &alpha1, Desc_Xs[o][k - JMP], d_X[k - JMP + 1], Desc_Ws[k - JMP], d_Ws[k - JMP], Conv_s_Desc, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM, d_ws, wsSize, &beta1, Desc_Ys[o][k - JMP], d_YY);
								d_t = d_YY;
							}

							if (PoolType[k] == 0)
							{								             
								BatchNormForward22 <BLOCKSIZE4> << <G[o].gridSizeBN2[k], blockSize4 >> >(d_ts, d_Y[k], d_t, d_F[k], d_SMUs[k], d_Param[k], OutCh[k], P[o].Yr[k]);
							}

							if (PoolType[k] == 1)
							{
								BatchNormForward22 <BLOCKSIZE4> << <G[o].gridSizeBN2[k], blockSize4 >> >(d_YY, d_Y[k], d_t, d_F[k], d_SMUs[k], d_Param[k], OutCh[k], P[o].Yr[k]);								         
							    MaxPoolingForward <1> << <G[o].gridSizeP[k], blockSize1 >> >(d_ts, d_YY, d_Indx[k], P[o].CR[k], P[o].SR[k], Sr1[k], Sr2[k], OutCh[k]);
							}

							if (PoolType[k] == 2)
							{
								             
								BatchNormForward22 <BLOCKSIZE4> << <G[o].gridSizeBN2[k], blockSize4 >> >(d_YY, d_Y[k], d_t, d_F[k], d_SMUs[k], d_Param[k], OutCh[k], P[o].Yr[k]);								            
								GlobalAvgPoolingForward<BLOCKSIZE3> << <gridSizePA, blockSize3 >> >(d_ts, d_YY, OutCh[k], P[o].CR[k] * P[o].CR[k]);
							}

						}
						else
						{
							if (PoolType[k] == 0)
							{								            
								BatchNormForward2 <BLOCKSIZE4> << <G[o].gridSizeBN2[k], blockSize4 >> >(d_ts, d_Y[k], d_SMUs[k], d_Param[k], OutCh[k], P[o].Yr[k]);
							}

							if (PoolType[k] == 1)
							{								            
								BatchNormForward2 <BLOCKSIZE4> << <G[o].gridSizeBN2[k], blockSize4 >> >(d_YY, d_Y[k], d_SMUs[k], d_Param[k], OutCh[k], P[o].Yr[k]);								        
								MaxPoolingForward<1> << <G[o].gridSizeP[k], blockSize1 >> >(d_ts, d_YY, d_Indx[k], P[o].CR[k], P[o].SR[k], Sr1[k], Sr2[k], OutCh[k]);
							}

							if (PoolType[k] == 2)
							{
								BatchNormForward2 <BLOCKSIZE4> << <G[o].gridSizeBN2[k], blockSize4 >> >(d_YY, d_Y[k], d_SMUs[k], d_Param[k], OutCh[k], P[o].Yr[k]);
								GlobalAvgPoolingForward<BLOCKSIZE3> << <gridSizePA, blockSize3 >> >(d_ts, d_YY, OutCh[k], P[o].CR[k] * P[o].CR[k]);
							}
						}
					}

					//=========================================================================================================================================

					blasstatus = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, Out1, BatchSize, In1 + 1, &alpha1, d_WF, Out1, d_Y0, In1 + 1, &beta1, d_YF, Out1);
					
					//-----------------------------------------------------------------------------------------------------------------------------------------

					//logsigInference<NumClasses, BLOCKSIZE2> << <Batch, BLOCKSIZE2 >> >((d_Yss + j*NumClasses), d_YF);
					SoftmaxInference<NumClasses, BLOCKSIZE2> << <BatchSize, BLOCKSIZE2 >> >((d_Yss + j*NumClasses), d_YF, (d_T + j), d_mse);				

					//=========================================================================================================================================

				}
			}
		}

		//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
		//
		//                                                              End of the inference Forloop. 
		//
		//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%

		PrintFinalResults(outResults, d_Yss, d_mse, d_T);

		hipFree(d_MTX); hipFree(d_Yss); hipFree(d_flip);
	}

	FreeRemainingMem(Red, Green, Blue, d_W, d_Ws, d_X, d_Y, d_Param, d_SMU, d_SMUs, d_Indx, d_F, d_WF, d_YF, d_Yv, d_YY, d_Y0, d_ws, d_mse, d_HeightTr, d_WidthTr, d_StartTr, d_T, PStart, StartPart, PartSize, TEMP);
	fclose(outResults);

	hipDeviceReset();

	//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
	//
	//                                                                          End of program
	//
	//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%

}
